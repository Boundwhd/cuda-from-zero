#include "hip/hip_runtime.h"
#include "../gemm.cuh"

__global__ void gemm_f32_f32_v1(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int M,
    const int N,
    const int K
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void launch_gemm_f32_f32_v1(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* __restrict__ C,
    const int M,
    const int N,
    const int K
) {
    dim3 block_size(32, 32);
    dim3 grid_size(CEIL(N, 32), CEIL(M, 32));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    gemm_f32_f32_v1<<<grid_size, block_size>>>(A, B, C, M, N, K);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel gemm_f32_f32_v1 execution time: %.3f ms\n", milliseconds / 10);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}